#include "hip/hip_runtime.h"
#include "gimage.h"
#include "array.h"
#include "timer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>

#define PI 3.14159265359
#define PRINT_INFO 1
#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

/**
* Template function for checking cuda errors. 
*/
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		exit(1);
	}
}

/**
* Helper function for cuda memory allocation. Simplifies calls to hipMalloc.
* @param d_p device pointer.
* @param size_t element the number of elements. 
*/
template<typename T>
hipError_t cudaAlloc(T*& d_p, size_t elements)
{
	return hipMalloc((void**)&d_p, elements * sizeof(T));
}

/**
* Helper function to select the proper CUDA device based on memory.
* @return int the device index to use.
*/
int selectDevice() {
	int devices;
	hipGetDeviceCount(&devices);
	if (devices > 1) {
		//need to select a device. 
		int bestDevice = -1;
		int maxMemory = -INT_MAX;
		for (int i = 0; i < devices; i++) {
			struct hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, i);
			int mem = properties.totalGlobalMem;
			if (mem > maxMemory) {
				maxMemory = mem;
				bestDevice = i;
			}
		}
		if (bestDevice != -1) {
			return bestDevice;
		}
		else {
			return 0;
		}
	}
	else {
		return 0;
	}
}

/**
* Confirms an RBG image to gray scale using the luminosity formula. 
* @param red the red channel
* @param green the green channel
* @param blue the blue channel.
* @param gray the gray scale output.
* @param numRows the number of rows in all the images.
* @param numCols the number of columns in all of the images.
*/
template<typename T>
__global__ void colorToGrey(T* red, T* green, T* blue, T* gray, int numRows, int numCols) {
	
	//get row and column in blcok
	int r = threadIdx.y + blockIdx.y*blockDim.y;
	int c = threadIdx.x + blockIdx.x*blockDim.x;
	//get unique point in image by finding position in grid.
	int index = c + r*blockDim.x*gridDim.x;
	int totalSize = numRows*numCols;
	if (index < totalSize) {
		double r = static_cast<double>(red[index]);
		double g = static_cast<double>(green[index]);
		double b = static_cast<double>(blue[index]);
		double grey = 0.21*r + 0.72*g + 0.07*b;
		gray[index] = static_cast<T>(grey);
	}
}

/**
* Gaussian blur kernal. Reads in a 16 bit image and outputs the blured image.
* @param d_in device input image.
* @param d_out device output image.
* @param filter gaussian filter array.
* @param numRows number of rows in the image.
* @param numCols number of cols in the image.
* @param blurSize the size of the blur.
*/
template<typename T>
__global__ void gaussian(T *d_in, T *d_out, const float* const filter, int numRows, int numCols, int blurSize) {
	//so filter width defines width of the filter.
	assert(blurSize % 2 == 1); //filter size should be odd.
	//get row and column in blcok
	int r = threadIdx.y + blockIdx.y*blockDim.y;
	int c = threadIdx.x + blockIdx.x*blockDim.x;
	//get unique point in image by finding position in grid.
	int offset = c + r*blockDim.x*gridDim.x;

	//holder for result of filtering. Hold it as a float for calculations. 
	float result = 0.0f;
	//check we don't access memory that doesn't exist. 
	if (offset >= numRows*numCols) {
		return;
	}
	//apply the filter. 
	for (int filter_r = -blurSize / 2; filter_r <= blurSize / 2; ++filter_r) {
		for (int filter_c = -blurSize / 2; filter_c <= blurSize / 2; ++filter_c) {
			//Find the global image position for this filter position
			//clamp to boundary of the image
			int rowCompare = r + filter_r >= 0 ? r + filter_r : 0;
			int colCompare = c + filter_c >= 0 ? c + filter_c : 0;
			//make sure we don't index rows and columns that don't exist. 
			int image_r = rowCompare <= static_cast<int>(numRows - 1) ? rowCompare : static_cast<int>(numRows - 1);
			int image_c = colCompare <= static_cast<int>(numCols - 1) ? colCompare : static_cast<int>(numCols - 1);

			float image_value = static_cast<float>(d_in[image_r * numCols + image_c]);
			float filter_value = filter[(filter_r + blurSize / 2) * blurSize + filter_c + blurSize / 2];
			//add filter value to result.
			result += image_value*filter_value;
		}
	}
	//set the output value. 
	d_out[offset] = static_cast<T>(result);
}


/**
* Generates a look up table used during window and leveling. 
* @param d_LUT device lookup table array. 
* @param window the window to use.
* @param level the level to use.
* @param levels the number of levels (this is also the size of d_LUT)
*/
__global__ void generateLUT(int* d_LUT, const int window, const int level, const int levels) {
	int a, b;
	int halfwin = window / 2;
	a = level - halfwin;
	if (a < 1) a = 1;
	b = level + halfwin;
	if (b > levels) b = levels;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < levels) {
		if (id < a) {
			d_LUT[id] = 0;
		}
		else if (id >= a && id <= b) {
			d_LUT[id] = ((levels / window)*(id - a));
		}
		else {
			d_LUT[id] = levels;
		}
	}
	
}

/**
* Performs window and leveling on a given image and stores result in the output. 
* @param input the input image. 
* @param output the output image. 
* @param d_LUT the device look up table. 
* @param numRows the number of rows in the image. 
* @param numCols the number of columns in the image. 
* @param window the window to use in the calculation.
* @param level the level to use in the calculation.
* @param levels the number of levels in the image and LUT
*/
template<typename T>
__global__ void cudaWindowLevel(T* input, T *output, int *d_LUT, int numRows, int numCols, int window, int level, int levels) {
	//get row and column in blcok
	int r = threadIdx.y + blockIdx.y*blockDim.y;
	int c = threadIdx.x + blockIdx.x*blockDim.x;
	//get unique point in image by finding position in grid.
	int index = c + r*blockDim.x*gridDim.x;
	if (index < numRows*numCols) {
		T in = input[index];
		if (in < levels) {
			int out = d_LUT[in];
			output[index] = (T)out;
		}	
	}
}

/**
* Calculates the gradient values and directions for a given input. Stores angles in d_theta and the gradient in d_gradient
* @param d_in the input image.
* @param d_gradient the array to store the gradient values.
* @param d_theta array to store our gradient directions. 
* @param k_gx the sobel operator in x (created on host)
* @param k_gy the sobel operator in y (created on host)
* @param numRows the number of rows in the image.
* @param numCols the number of columns in the image. 
*/
template<typename T>
__global__ void gradientAndDirection(T *d_in, T *d_gradient, int* d_theta, int* k_gx, int* k_gy, int numRows, int numCols) {
	//get row and column in the current grid (this should be a sub set of the image if it is large enough.
	int r = threadIdx.y + blockIdx.y*blockDim.y;
	int c = threadIdx.x + blockIdx.x*blockDim.x;
	//get unique point in image by finding position in grid.
	int index = c + r*blockDim.x*gridDim.x;

	if (index >= numRows*numCols) {
		return;
	}

	//run convolution on the image with the sobel filters. 
	float x_res = 0.0f;
	float y_res = 0.0f;
	int kernelSize = 3;
	//apply the filter. 
	for (int filter_r = -kernelSize / 2; filter_r <= kernelSize / 2; ++filter_r) {
		for (int filter_c = -kernelSize / 2; filter_c <= kernelSize / 2; ++filter_c) {
			//Find the global image position for this filter position
			//clamp to boundary of the image
			int rowCompare = r + filter_r >= 0 ? r + filter_r : 0;
			int colCompare = c + filter_c >= 0 ? c + filter_c : 0;
			//make sure we don't index rows and columns that don't exist. 
			int image_r = rowCompare <= static_cast<int>(numRows - 1) ? rowCompare : static_cast<int>(numRows - 1);
			int image_c = colCompare <= static_cast<int>(numCols - 1) ? colCompare : static_cast<int>(numCols - 1);

			float image_value = static_cast<float>(d_in[image_r * numCols + image_c]);
			float filter_x = static_cast<float>(k_gx[(filter_r + kernelSize / 2) * kernelSize + filter_c + kernelSize / 2]);
			float filter_y = static_cast<float>(k_gy[(filter_r + kernelSize / 2) * kernelSize + filter_c + kernelSize / 2]);
			//add filter value to result.
			x_res += image_value*filter_x;
			y_res += image_value*filter_y;
		}
	}

	//store the gradient magnitude. 
	d_gradient[index] = static_cast<T>(sqrtf(powf(x_res, 2.0f) + powf(y_res, 2.0f)));
	double angle = (atan2f(y_res, x_res)) / PI * 180.0;
	int correctAngle;
	/* Convert actual edge direction to approximate value */
	if (((angle < 22.5) && (angle > -22.5)) || (angle > 157.5) || (angle < -157.5))
		correctAngle = 0;
	if (((angle > 22.5) && (angle < 67.5)) || ((angle < -112.5) && (angle > -157.5)))
		correctAngle = 45;
	if (((angle > 67.5) && (angle < 112.5)) || ((angle < -67.5) && (angle > -112.5)))
		correctAngle = 90;
	if (((angle > 112.5) && (angle < 157.5)) || ((angle < -22.5) && (angle > -67.5)))
		correctAngle = 135;
	//store the angle. 
	d_theta[index] = correctAngle;
}

/**
* Performs non-maximum suppression to further isolate edges during canny edge detection. This will save edges that are local maxima. It will then check 
* if the value is greater than the upper threshold, between the upper and lower threshold or below the lower threshold. If it is above the upper threshold the value
* is labeled as definitely being part of an edge. If it is inbetween, it will only be counted as an edge if it is connected to a definite edge, and if it below the lower
* threhold, it will be discarded.
* @param d_gradMag device pointer to gradient magnitude array.
* @param d_theta device pointer to gradient direction array.
* @param d_out the output array with the final edges. 
* @param upperThresh the upper threshold to check against.
* @param lowerThresh the lower threshold to check against. 
* @param numRows the number of rows in the arrays (should be the same for all).
* @param numCols the number of columns in the arrays (should be the same for all).
*/
template<typename T>
__global__ void nonMaximumSuppression(T* d_gradMag, int* d_theta, T* d_out, int numRows, int numCols) {
	//get row and column in the current grid (this should be a sub set of the image if it is large enough.
	int r = threadIdx.x + blockIdx.x*blockDim.x;
	int c = threadIdx.y + blockIdx.y*blockDim.y;
	//get unique point in image by finding position in grid.
	int index = c + r*blockDim.x*gridDim.x;

	if (index >= numRows*numCols) {
		return;
	}

	T value = d_gradMag[index];
	int direction = d_theta[index];
	int fCheck, sCheck = -1;
	switch (direction) {
	case 0:
		//horizontal
		fCheck = r + (c-1)*blockDim.x*gridDim.x;
		sCheck = r + (c+1)*blockDim.x*gridDim.x;
		break;
	case 45:
		//one row less one column more
		fCheck = (r - 1) + (c+1)*blockDim.x*gridDim.x;
		sCheck = (r + 1) + (c-1)*blockDim.x*gridDim.x;
		break;
	case 90:
		//vertical
		fCheck = (r - 1) + c*blockDim.x*gridDim.x;
		sCheck = (r + 1) + c*blockDim.x*gridDim.x;
		break;
	case 135:
		fCheck = (r - 1) + (c -1)*blockDim.x*gridDim.x;
		sCheck = (r + 1) + (c + 1)*blockDim.x*gridDim.x;
		break;
	}

	if (fCheck < numRows*numCols && sCheck < numRows*numCols) {
		T v1 = d_gradMag[fCheck];
		T v2 = d_gradMag[sCheck];
		int maxIndex = -1;
		if (value > v1 && value > v2) {
			maxIndex = index;
		}
		else if (value < v1 && v1 > v2) {
			maxIndex = fCheck;
		}
		else if (value < v2 && v2 > v1){
			maxIndex = sCheck;
		}

		if (maxIndex > 0 && maxIndex < numRows*numCols) {
			d_out[maxIndex] = d_gradMag[maxIndex];	
		}
	}
}

/**
* Performs hysteresis thresholding using two thresholds. If a value is above the upper threshold then it is deemed to definitely be an edge.
* If it is between the two thresholds, then it is only considered an edge if it is connected to a definite edge. If it is below or eqaul to 
* the lower threshold then it is definitely not an edge. 
* @param d_in input gradient magnitudes after non maximum suppression.
* @param d_out output edge image.
* @param theta gradient directions
* @param upper the upper threshold
* @param lower the lower threshold
* @param numRows the number of rows in all the arrays
* @param numCols the number of columns in all the arrays
*/
template<typename T> 
__global__ void hysteresisThresholding(T* d_in, T* d_out, int* theta, int upper, int lower, int numRows, int numCols) {
	//get row and column in the current grid (this should be a sub set of the image if it is large enough.
	int r = threadIdx.x + blockIdx.x*blockDim.x;
	int c = threadIdx.y + blockIdx.y*blockDim.y;
	//get unique point in image by finding position in grid.
	int index = c + r*blockDim.x*gridDim.x;
	int totalSize = numRows*numCols;
	if (index < totalSize) {
		T value = d_in[index];
		if (static_cast<int>(value) >= upper) {
			d_out[index] = value;
		}
		else if (static_cast<int>(value) <= lower) {
			d_out[index] = static_cast<T>(0);
		}
		else {
			//inbetween both values so walk the path. 
			//get the direction.
			int totBlockSize = blockDim.x*gridDim.x;
			int direction = theta[index];
			int rowOffset, colOffset = 1;
			bool foundFirst = false;
			while (true) {
				//traverse the path.
				int idxOne = -1;
				switch (direction) {
				case 0:
					//traverse column wise. 
					idxOne = r + (c - colOffset)*totBlockSize;
					break;
				case 45:
					idxOne = (r - rowOffset) + (c + colOffset)*totBlockSize;
					break;
				case 90:
					idxOne = (r - rowOffset) + c*totBlockSize;
					break;
				case 135:
					idxOne = (r - rowOffset) + (c - colOffset)*totBlockSize;
					break;
				}

				if (idxOne < totalSize && idxOne >= 0) {
					T v1 = d_in[idxOne];
					int dirOne = theta[idxOne];
					int v1Cast = static_cast<int>(v1);

					if (v1Cast <= lower) {
						//below lower threshold so no good. 
						foundFirst = false;
						break;
					}
					else if (v1Cast >= upper) {
						//def an edge so we're good. 
						foundFirst = true;
						//go ahead and set the value.
						d_out[index] = value;
						break;
					}
					else if (dirOne != direction) {
						foundFirst = false;
						break;
					}
					else {
						//increment and continue.
						colOffset++;
						rowOffset++;
					}
				}
				else {
					if (r + rowOffset >= numRows || c + colOffset >= numCols) {
						foundFirst = false;
						break;
					}
				}
				

			}

			//reset offsets. 
			colOffset = 1;
			rowOffset = 1;

			if (!foundFirst) {
				while (true) {
					int idxTwo = -1;
					switch (direction) {
					case 0:
						//traverse column wise. 
						idxTwo = r + (c + colOffset)*totBlockSize;
						break;
					case 45:
						idxTwo = (r + rowOffset) + (c - colOffset)*totBlockSize;
						break;
					case 90:
						idxTwo = (r + rowOffset) + c*totBlockSize;
						break;
					case 135:
						idxTwo = (r + rowOffset) + (c + colOffset)*totBlockSize;
						break;
					}

					if (idxTwo < totalSize && idxTwo >= 0) {

						T v2 = d_in[idxTwo];
						int dirTwo = theta[idxTwo];
						int v2Cast = static_cast<int>(v2);

						if (v2Cast <= lower) {
							//below lower threshold so no good. 
							d_out[index] = static_cast<T>(0);
							break;
						}
						else if (v2Cast >= upper) {
							//def an edge so we're good. 
							d_out[index] = value;
							break;
						}
						else if (dirTwo != direction) {
							d_out[index] = static_cast<T>(0);
							break;
						}
						else {
							//increment and continue.
							colOffset++;
							rowOffset++;
						}
					}
					else {
						if (r + rowOffset >= numRows || c + colOffset >= numCols) {
							d_out[index] = static_cast<T>(0);
						}
					}
				}
			}	
		}
	}
}

template<typename T>
__global__ void houghCircles(T* d_in, T* d_accumalator, int radius, int numRows, int numCols) {
	//get row and column in the current grid (this should be a sub set of the image if it is large enough.
	int r = threadIdx.x + blockIdx.x*blockDim.x;
	int c = threadIdx.y + blockIdx.y*blockDim.y;
	//get unique point in image by finding position in grid.
	int index = c + r*blockDim.x*gridDim.x;
	int totalSize = numRows*numCols;
	if (index < totalSize) {
		//TODO: Finish implementing hough circles, should go through all the 
		//angles for a circle (i.e. 0 to 2pi) and then get a and b. Save these
		//"votes" in the accumulator matrix. Will find the maximums of this accumulator
		//matrix later. 
	}
}

/**
* Namespace for all gimage functions.
*/
namespace gimage {

	Array& Array::operator=(Array& other) {
		if (this == &other) {
			return *this;
		}
		return *this;
	}

	DoubleArray::DoubleArray(int rows, int cols) : Array(rows, cols, Type::DOUBLE) {
		allocate(size());
	}

	DoubleArray::DoubleArray(DoubleArray &other) : Array(other.rows, other.cols, Type::DOUBLE){
		allocate(other.size());
		double* oData = static_cast<double*>(other.hostData());
		std::memcpy(h_data, oData, totalSize());
	}

	DoubleArray::~DoubleArray() {
		if (h_data) {
			delete[] h_data;
		}
		if (d_data) {
			checkCudaErrors(hipFree(d_data));
			d_data = NULL;
		}
	}

	Array& DoubleArray::operator+(Array &other) {
		assert(rows == other.rows && cols == other.cols);
		DoubleArray out(rows, cols);
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				out.setData<double>(r, c, at<double>(r, c) +
					other.at<double>(r, c));
			}
		}

		return out;
	}

	Array& DoubleArray::operator-(Array &other) {
		assert(rows == other.rows && cols == other.cols);
		DoubleArray out(rows, cols);
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				out.setData<double>(r, c, at<double>(r, c) -
					other.at<double>(r, c));
			}
		}

		return out;
	}

	Array& DoubleArray::operator=(Array &other) {
		if (this == &other) {
			return *this;
		}
		assert(other.getType() == getType());
		rows = other.rows;
		cols = other.cols;
		setSize(rows*cols);
		delete[] h_data;
		allocate(totalSize());
		double* otherData = static_cast<double*>(other.hostData());
		std::memcpy(h_data, otherData, totalSize());
		return *this;
	}

	void* DoubleArray::hostData() {
		return h_data;
	}

	void* DoubleArray::deviceData() {
		return d_data;
	}

	
	void DoubleArray::gpuAlloc() {
		if (!d_data) {
			checkCudaErrors(cudaAlloc(d_data, size()));
		}
	}

	void DoubleArray::gpuFree() {
		if (d_data) {
			checkCudaErrors(hipFree(d_data));
			d_data = NULL;
		}	
	}

	int DoubleArray::totalSize() {
		return size() * sizeof(double);
	}

	void DoubleArray::clone(Array& other) {
		assert(other.getType() == getType());
		assert(other.size() == size());
		std::memcpy(static_cast<double*>(other.hostData()), 
			static_cast<double*>(hostData()), totalSize());
	}

	void DoubleArray::memcpy(MemcpyDirection dir) {
		if (dir == MemcpyDirection::HOST_TO_DEVICE) {
			checkCudaErrors(hipMemcpy(d_data, h_data, totalSize(), hipMemcpyHostToDevice));
		}
		else {
			checkCudaErrors(hipMemcpy(h_data, d_data, totalSize(), hipMemcpyDeviceToHost));
		}
	}

	void DoubleArray::allocate(int size) {
		h_data = new double[size];
	}

	ArrayUint16::ArrayUint16(int rows, int cols) : Array(rows, cols, Type::UINT16) {
		allocate(size());
	}

	ArrayUint16::ArrayUint16(ArrayUint16 &other) : Array(other.rows, other.cols, Type::UINT16){
		allocate(other.size());
		uint16_t* oData = static_cast<uint16_t*>(other.hostData());
		std::memcpy(h_data, oData, totalSize());
	}

	ArrayUint16::~ArrayUint16() {
		delete[] h_data;
		if (d_data) {
			checkCudaErrors(hipFree(d_data));
			d_data = NULL;
		}
	}

	Array& ArrayUint16::operator=(Array &other) {
		if (this == &other) {
			return *this;
		}
		assert(other.getType() == getType());
		rows = other.rows;
		cols = other.cols;
		setSize(rows*cols);
		delete[] h_data;
		allocate(totalSize());
		uint16_t* otherData = static_cast<uint16_t*>(other.hostData());
		std::memcpy(h_data, otherData, totalSize());
		return *this;
	}

	Array& ArrayUint16::operator+(Array &other) {
		assert(rows == other.rows && cols == other.cols);
		ArrayUint16 out(rows, cols);
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				out.setData<uint16_t>(r, c, at<uint16_t>(r, c) +
					other.at<uint16_t>(r, c));
			}
		}

		return out;
	}

	Array& ArrayUint16::operator-(Array &other) {
		assert(rows == other.rows && cols == other.cols);
		ArrayUint16 out(rows, cols);
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				out.setData<uint16_t>(r, c, at<uint16_t>(r, c) -
					other.at<uint16_t>(r, c));
			}
		}

		return out;
	}

	void* ArrayUint16::hostData() {
		return h_data;
	}

	void* ArrayUint16::deviceData() {
		return d_data;
	}

	void ArrayUint16::gpuAlloc() {
		if (!d_data) {
			checkCudaErrors(cudaAlloc(d_data, size()));
		}
	}

	void ArrayUint16::gpuFree() {
		if (d_data) {
			checkCudaErrors(hipFree(d_data));
			d_data = NULL;
		}	
	}

	int ArrayUint16::totalSize() {
		return size() * sizeof(uint16_t);
	}

	void ArrayUint16::memcpy(MemcpyDirection dir) {
		if (dir == MemcpyDirection::HOST_TO_DEVICE) {
			checkCudaErrors(hipMemcpy(d_data, h_data, totalSize(), hipMemcpyHostToDevice));
		}
		else {
			checkCudaErrors(hipMemcpy(h_data, d_data, totalSize(), hipMemcpyDeviceToHost));
		}
	}

	void ArrayUint16::clone(Array& other) {
		assert(other.getType() == getType());
		assert(other.size() == size());
		std::memcpy(static_cast<uint16_t*>(other.hostData()), 
			static_cast<uint16_t*>(hostData()), totalSize());
	}

	void ArrayUint16::allocate(int size) {
		h_data = new uint16_t[size];
	}

	ArrayUint8::ArrayUint8(int rows, int cols) : Array(rows, cols, Type::UINT8) {
		allocate(size());
	}

	ArrayUint8::ArrayUint8(ArrayUint8 &other) : Array(other.rows, other.cols, Type::UINT8){
		allocate(other.size());
		uint8_t* oData = static_cast<uint8_t*>(other.hostData());
		std::memcpy(h_data, oData, totalSize());
	}

	ArrayUint8::~ArrayUint8() {
		delete[] h_data;
		if (d_data) {
			checkCudaErrors(hipFree(d_data));
			d_data = NULL;
		}
	}

	Array& ArrayUint8::operator+(Array &other) {
		assert(rows == other.rows && cols == other.cols);
		ArrayUint8 out(rows, cols);
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				out.setData<uint8_t>(r, c, at<uint8_t>(r, c) +
					other.at<uint8_t>(r, c));
			}
		}

		return out;
	}

	Array& ArrayUint8::operator-(Array &other) {
		assert(rows == other.rows && cols == other.cols);
		ArrayUint8 out(rows, cols);
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				out.setData<uint8_t>(r, c, at<uint8_t>(r, c) -
					other.at<uint8_t>(r, c));
			}
		}

		return out;
	}

	Array& ArrayUint8::operator=(Array &other) {
		if (this == &other) {
			return *this;
		}
		assert(other.getType() == getType());
		rows = other.rows;
		cols = other.cols;
		setSize(rows*cols);
		delete[] h_data;
		allocate(totalSize());
		uint8_t* otherData = static_cast<uint8_t*>(other.hostData());
		std::memcpy(h_data, otherData, totalSize());
		return *this;
	}

	void* ArrayUint8::hostData() {
		return h_data;
	}

	void* ArrayUint8::deviceData() {
		return d_data;
	}

	void ArrayUint8::gpuAlloc() {
		if (!d_data) {
			checkCudaErrors(cudaAlloc(d_data, size()));
		}
	}

	void ArrayUint8::gpuFree() {
		if (d_data) {
			checkCudaErrors(hipFree(d_data));
			d_data = NULL;
		}
	}

	int ArrayUint8::totalSize() {
		return size() * sizeof(uint8_t);
	}

	void ArrayUint8::memcpy(MemcpyDirection dir) {
		if (dir == MemcpyDirection::HOST_TO_DEVICE) {
			checkCudaErrors(hipMemcpy(d_data, h_data, totalSize(), hipMemcpyHostToDevice));
		}
		else {
			checkCudaErrors(hipMemcpy(h_data, d_data, totalSize(), hipMemcpyDeviceToHost));
		}
	}

	void ArrayUint8::clone(Array& other) {
		assert(other.getType() == getType());
		assert(other.size() == size());
		std::memcpy(static_cast<uint8_t*>(other.hostData()),
			static_cast<uint8_t*>(hostData()), totalSize());
	}

	void ArrayUint8::allocate(int size) {
		h_data = new uint8_t[size];
	}

	MatrixD::MatrixD(int size) : DoubleArray(1, size) {
	}

	MatrixD::MatrixD(int rows, int cols) : DoubleArray(rows, cols) {
	}

	double MatrixD::det() {
		//TODO: calculate determinant
		return 1.0;
	}

	/**
	* Performs matrix multiplication. 
	*/
	MatrixD MatrixD::operator*(MatrixD other) {
		assert(cols == other.rows);
		MatrixD out(rows, other.cols);
	
		int outRows = out.rows;
		int outCols = out.cols;
		for (int r = 0; r < outRows; r++) {
			for (int c = 0; c < outCols; c++) {
				//find sum for this position. 
				double sum = 0.0;
				for (int i = 0; i < other.rows; i++) {
					sum += other.at<double>(i, c) * at<double>(r, i);
				}
				out.setData<double>(r, c, sum);
			}
		}
		return out;
	}

	/**
	* Converts a color image to a grayscale image.
	* @param
	*/
	void GIMAGE_EXPORT rgbToGray(ArrayUint8& red, ArrayUint8& green, ArrayUint8& blue, ArrayUint8& gray) {

		//allocate arrays and move data to device. 
		red.gpuAlloc();
		green.gpuAlloc();
		blue.gpuAlloc();
		gray.gpuAlloc();
		
		//move data to device.
		red.memcpy(MemcpyDirection::HOST_TO_DEVICE);
		green.memcpy(MemcpyDirection::HOST_TO_DEVICE);
		blue.memcpy(MemcpyDirection::HOST_TO_DEVICE);

		//set the gray image to nothing.
		checkCudaErrors(hipMemset(static_cast<uint8_t*>(gray.deviceData()), 0, gray.totalSize()));
		int numRows = red.rows;
		int numCols = red.cols;

		//select the device
		int device = selectDevice();
		checkCudaErrors(hipSetDevice(device));
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		//get max threads and threads per block.
		int maxThreadsPerBlock = properties.maxThreadsPerBlock;
		int threadsPerBlock = std::sqrt(maxThreadsPerBlock);
#if PRINT_INFO
		std::cout << "GPU: " << properties.name << std::endl;
		std::cout << "Using " << properties.multiProcessorCount << " multiprocessors" << std::endl;
		std::cout << "Max threads per block: " << properties.maxThreadsPerBlock << std::endl;
		std::cout << "Max grid size: " << properties.maxGridSize[0] << std::endl;
		std::cout << "Threads per block " << threadsPerBlock << std::endl;
#endif	

		//specify block size. 
		dim3 block_size(threadsPerBlock, threadsPerBlock);
		/*
		* Specify the grid size for the GPU.
		* Make it generalized, so that the size of grid changes according to the input image size
		*/
		dim3 grid_size;
		grid_size.x = (numCols + block_size.x - 1) / block_size.x;  /*< Greater than or equal to image width */
		grid_size.y = (numRows + block_size.y - 1) / block_size.y; /*< Greater than or equal to image height */
		
		//gpu timer for 
		GpuTimer timer;
		timer.Start();
		colorToGrey << <grid_size, block_size >> >(static_cast<uint8_t*>(red.deviceData()), static_cast<uint8_t*>(green.deviceData()),
			static_cast<uint8_t*>(blue.deviceData()), static_cast<uint8_t*>(gray.deviceData()), numRows, numCols);
		timer.Stop();
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError()); 
		//measure how long the kernel took
		float ms = timer.Elapsed();

#if PRINT_INFO
		printf("RGB to gray kernel took %f ms.\n", ms);
#endif
		//copy results back.
		gray.memcpy(MemcpyDirection::DEVICE_TO_HOST);

		//free gpu memory. 
		red.gpuFree();
		green.gpuFree();
		blue.gpuFree();
		gray.gpuFree();
	}

	/**
	* Performs a Gaussian blur on a given image and stores it in the output.
	* @param input the input image
	* @param output the output image.
	* @param numRows the number of rows in the input image.
	* @param numCols the number of columns int he input image.
	* @param blurSize the size of the blur. This must be odd. Note that the blur filter will be square.
	*/
	void GIMAGE_EXPORT gaussianBlur(Array& input, Array& output, float sigma, int numRows, int numCols, int blurSize) {
		//blur size must be odd. 
		assert(blurSize % 2 == 1);
		//first calculate the filter. 
		float *h_filter = new float[blurSize*blurSize];
		float filterSum = 0.f;

		for (int r = -blurSize / 2; r <= blurSize / 2; ++r) {
			for (int c = -blurSize / 2; c <= blurSize / 2; ++c) {
				float filterValue = expf(-(float)(c * c + r * r) / (2.f * sigma * sigma));
				h_filter[(r + blurSize / 2) * blurSize + c + blurSize / 2] = filterValue;
				filterSum += filterValue;
			}
		}

		float normalizationFactor = 1.f / filterSum;

		for (int r = -blurSize / 2; r <= blurSize / 2; ++r) {
			for (int c = -blurSize / 2; c <= blurSize / 2; ++c) {
				h_filter[(r + blurSize / 2) * blurSize + c + blurSize / 2] *= normalizationFactor;
			}
		}

		//select the device
		int device = selectDevice();
		checkCudaErrors(hipSetDevice(device));
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);

		//now we can filter the image. 
		int size = input.totalSize();
		float *d_filter;
		int totalBlurSize = blurSize*blurSize;

		//allocated memory for filter. 
		checkCudaErrors(cudaAlloc(d_filter, totalBlurSize));
		checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float)*totalBlurSize, hipMemcpyHostToDevice));

		//allocate image memory.
		input.gpuAlloc();
		output.gpuAlloc();
		//copy memory to device. 
		input.memcpy(MemcpyDirection::HOST_TO_DEVICE);

		int maxThreadsPerBlock = properties.maxThreadsPerBlock;
		int threadsPerBlock = std::sqrt(maxThreadsPerBlock);
#if PRINT_INFO
		std::cout << "GPU: " << properties.name << std::endl;
		std::cout << "Using " << properties.multiProcessorCount << " multiprocessors" << std::endl;
		std::cout << "Max threads per block: " << properties.maxThreadsPerBlock << std::endl;
		std::cout << "Max grid size: " << properties.maxGridSize[0] << std::endl;
		std::cout << "Threads per block " << threadsPerBlock << std::endl;
#endif		
		//specify block size. 
		dim3 block_size(threadsPerBlock, threadsPerBlock);
		/*
		* Specify the grid size for the GPU.
		* Make it generalized, so that the size of grid changes according to the input image size
		*/
		dim3 grid_size;
		grid_size.x = (numCols + block_size.x - 1) / block_size.x;  /*< Greater than or equal to image width */
		grid_size.y = (numRows + block_size.y - 1) / block_size.y; /*< Greater than or equal to image height */
#if PRINT_INFO
		std::cout << "Grid size: (" << grid_size.x << " , " << grid_size.y << ")" << std::endl;
#endif	
		//get image type.
		gimage::Type t = input.getType();
		GpuTimer timer;
		
		switch (t) {
		case Type::UINT16:	
			//call the kernal.
			timer.Start();
			gaussian << <grid_size, block_size >> >(static_cast<uint16_t*>(input.deviceData()), static_cast<uint16_t*>(output.deviceData()), 
				d_filter, numRows, numCols, blurSize);
			timer.Stop();
			break;
		case Type::DOUBLE:
			//call the kernal.
			timer.Start();
			gaussian << <grid_size, block_size >> >(static_cast<double*>(input.deviceData()), static_cast<double*>(output.deviceData()),
				d_filter, numRows, numCols, blurSize);
			timer.Stop();
			break;
		}

		float ms = timer.Elapsed();
#if PRINT_INFO
		printf("Kernel took %f ms\n", ms);
#endif
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		output.memcpy(MemcpyDirection::DEVICE_TO_HOST);

		//clean up gpu
		output.gpuFree();
		input.gpuFree();
		checkCudaErrors(hipFree(d_filter));

		//clean up host
		delete[] h_filter;	
	}

	/**
	* Performs the look up table method of window and leveling on the given image and stores the result in out.
	* @param input the input image.
	* @param out the output image.
	* @param numRows the number of rows in the image.
	* @param numCols the number of columns in the image.
	* @param window the window to use in the calculation.
	* @param level the level to use in the calculation.
	*/
	void GIMAGE_EXPORT windowAndLevel(Array& input, Array& out, int numRows, int numCols, int window, int level) {
		
		//perform assertions
		assert(input.getType() == out.getType());
		assert(input.size() == out.size());
		assert(input.rows == out.rows && input.cols == out.cols && input.rows == numRows && input.cols == numCols);
		assert(window > 0 && level > 0);

		//select the device
		int device = selectDevice();
		checkCudaErrors(hipSetDevice(device));
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		int maxThreadsPerBlock = properties.maxThreadsPerBlock;
		int threadsPerBlock = std::sqrt(maxThreadsPerBlock);

		//get number of levels. 
		int levels = (1 << 16) - 1;
		int* d_LUT;
		
		checkCudaErrors(cudaAlloc(d_LUT, levels));
		checkCudaErrors(hipMemset(d_LUT, 0, sizeof(int)*levels));

		int lutBlocks = levels / maxThreadsPerBlock;

		//generate the look up table. 
		GpuTimer timer;
		timer.Start();
		generateLUT << <lutBlocks, maxThreadsPerBlock >> > (d_LUT, window, level, levels);
		timer.Stop();
		float msLut = timer.Elapsed();
#if PRINT_INFO
		printf("LUT calc used %d blocks\n", lutBlocks);
		printf("LUT kernel took %f ms\n", msLut);
#endif		
		//specify block size. 
		dim3 block_size(threadsPerBlock, threadsPerBlock);
		/*
		* Specify the grid size for the GPU.
		* Make it generalized, so that the size of grid changes according to the input image size
		*/
		dim3 grid_size;
		grid_size.x = (numCols + block_size.x - 1) / block_size.x;  /*< Greater than or equal to image width */
		grid_size.y = (numRows + block_size.y - 1) / block_size.y; /*< Greater than or equal to image height */

		//allocate device memory.
		input.gpuAlloc();
		out.gpuAlloc();
		//copy host memory to device. 
		input.memcpy(MemcpyDirection::HOST_TO_DEVICE);

		//initialize the timer. 
		GpuTimer winT;

		gimage::Type t = input.getType();
		switch (t) {
			case Type::UINT16:
				//now actually apply the window and leveling. 
				winT.Start();
				cudaWindowLevel << <grid_size, block_size >> >(static_cast<uint16_t*>(input.deviceData()), 
																static_cast<uint16_t*>(out.deviceData()), 
																d_LUT, numRows, numCols, window, level, levels);
				winT.Stop();
				break;
		}

		float ms = winT.Elapsed();
#if PRINT_INFO
		printf("WindowLevel kernel took %f ms\n", ms);
#endif
		//copy back data. 
		out.memcpy(MemcpyDirection::DEVICE_TO_HOST);

		//clean up. 
		input.gpuFree();
		out.gpuFree();
		checkCudaErrors(hipFree(d_LUT));
	}

	/**
	* Performs canny edge detection on the input and outputs an image with only the edges in the output. 
	* This performs the following steps: (1) Apply a gaussian filter to smooth the image.
	* (2) Apply the Sobel operator in the x and y directions and keep track of gradient/direction.
	* (3) Perform non maximum suppression on the image.
	* (4) Use dual hysteresis thresholding to further eliminate false edges. 
	*/
	void GIMAGE_EXPORT cannyEdgeDetector(Array& input, Array& output, int numRows, int numCols,
		float sigma, int lowerThresh, int upperThresh) {

		assert(input.getType() == output.getType());
		assert(sigma > 0);
		assert(lowerThresh < upperThresh);
		assert(lowerThresh > 0 && upperThresh > 0);

		//set the device. 
		int device = selectDevice();
		checkCudaErrors(hipSetDevice(device));
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);

		//calculate the threds per block. 
		int maxThreadsPerBlock = properties.maxThreadsPerBlock;
		int threadsPerBlock = std::sqrt(maxThreadsPerBlock);

		//specify block size. 
		dim3 block_size(threadsPerBlock, threadsPerBlock);
		/*
		* Specify the grid size for the GPU.
		* Make it generalized, so that the size of grid changes according to the input image size
		*/
		dim3 grid_size;
		grid_size.x = (numCols + block_size.x - 1) / block_size.x;  /*< Greater than or equal to image width */
		grid_size.y = (numRows + block_size.y - 1) / block_size.y; /*< Greater than or equal to image height */

	
		//create Sobel kernels
		int *k_gx = new int[9];
		int *k_gy = new int[9];
		k_gx[0] = -1; k_gx[1] = 0; k_gx[2] = 1;
		k_gx[3] = -2; k_gx[4] = 0; k_gx[5] = 2;
		k_gx[6] = -1; k_gx[7] = 0; k_gx[8] = 1;

		k_gy[0] = 1; k_gy[1] = 2; k_gy[2] = 1;
		k_gy[3] = 0; k_gy[4] = 0; k_gy[5] = 0;
		k_gy[6] = -1; k_gy[7] = -2; k_gy[8] = -1;

		//create device copies of the sobel kernels.
		int* d_kgx;
		int *d_kgy;
		checkCudaErrors(cudaAlloc(d_kgx, 9));
		checkCudaErrors(cudaAlloc(d_kgy, 9));
		checkCudaErrors(hipMemcpy(d_kgx, k_gx, sizeof(int) * 9, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_kgy, k_gy, sizeof(int) * 9, hipMemcpyHostToDevice));

		int *d_theta;
		checkCudaErrors(cudaAlloc(d_theta, input.size()));

		//check the image type.
		gimage::Type t = input.getType();
		switch (t) {
			case Type::UINT16:
				gimage::ArrayUint16 blurred(numRows, numCols);
				//run gaussian blur first. 
				gaussianBlur(input, blurred, sigma, numRows, numCols, 5);
				uint16_t *d_gradient;
				
				//allocate all our arrays. 
				checkCudaErrors(cudaAlloc(d_gradient, input.size()));
		
				//allocate on gpu. 
				input.gpuAlloc();
				//copy data to GPU.
				input.memcpy(MemcpyDirection::HOST_TO_DEVICE);

				uint16_t* d_in;
				uint16_t* d_out;
				//get input device pointer. 
				d_in = static_cast<uint16_t*>(input.deviceData());
				
				GpuTimer timer;
				timer.Start();
				//call our gradient kernel
				gradientAndDirection << <grid_size, block_size >> >(d_in, d_gradient, d_theta, d_kgx, d_kgy, numRows, numCols);
				timer.Stop();
				float gradMs = timer.Elapsed();
#if PRINT_INFO
				printf("Gradient kernel took %f ms\n", gradMs);
#endif
				//synchronize the device. 
				hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

				//allocated output.
				output.gpuAlloc();
				d_out = static_cast<uint16_t*>(output.deviceData());

				//create non maximum suppression output.
				ArrayUint16 nonMaxOut(output.rows, output.cols);
				//allocate on GPU.
				nonMaxOut.gpuAlloc();
				//get device pointer.
				uint16_t* d_nMax_out = static_cast<uint16_t*>(nonMaxOut.deviceData());
				
				//set output to zeros.
				checkCudaErrors(hipMemset(d_out, 0, output.totalSize()));
				timer.Start();
				//perform non maximum suppression. 
				nonMaximumSuppression << <grid_size, block_size >> >(d_gradient, d_theta, d_nMax_out, numRows, numCols);
				timer.Stop();
				float nonMaxMs = timer.Elapsed();
#if PRINT_INFO
				printf("Non-maximum kernel took %f ms\n", nonMaxMs);
#endif
				hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

				timer.Start();
				hysteresisThresholding << <grid_size, block_size >> >(d_nMax_out, d_out, d_theta, upperThresh, lowerThresh, numRows, numCols);
				timer.Stop();
				float hysMs = timer.Elapsed();
#if PRINT_INFO
				printf("Thresholding kernel took %f ms\n", hysMs);
#endif
				hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

				//copy result to gpu. 
				checkCudaErrors(hipMemcpy(static_cast<uint16_t*>(output.hostData()), d_out, output.totalSize(), hipMemcpyDeviceToHost));

				//free up used memory. 
				input.gpuFree();
				output.gpuFree();
				nonMaxOut.gpuFree();

				checkCudaErrors(hipFree(d_gradient));
				checkCudaErrors(hipFree(d_theta));
				break;
		}
		
		//free our gpu filters. 
		checkCudaErrors(hipFree(d_kgx));
		checkCudaErrors(hipFree(d_kgy));

		//free cpu memory. 
		delete[] k_gx;
		delete[] k_gy;
	}
}

